#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "DS_timer.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define NUM_DATA 8192

// problem 1
__global__ void vecMulDiv(double *a, double *b, double *c, double *d, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        d[idx] = a[idx] * b[idx] / c[idx];
    }
}
int main(void)
{
    double *a, *b, *c, *d, *hd;
    double *da, *db, *dc, *dd; 
    int memsize = sizeof(double) * NUM_DATA;
    int result;

    DS_timer timer(5);
    timer.setTimerName(0, (char *)"CUDA Total");
    timer.setTimerName(1, (char *)"Computation on device (GPU)");
    timer.setTimerName(2, (char *)"Memory copy: host -> device");
    timer.setTimerName(3, (char *)"Memory copy: device -> host");
    timer.setTimerName(4, (char *)"Computation on host (CPU)");

    // memory allocation on host
    a = (double *)malloc(memsize);
    b = (double *)malloc(memsize);
    c = (double *)malloc(memsize);
    d = (double *)malloc(memsize);
    hd = (double *)malloc(memsize);

    // initialize
    memset(a, 0, memsize);
    memset(b, 0, memsize);
    memset(c, 0, memsize);
    memset(d, 0, memsize);
    memset(hd, 0, memsize);

    // set data
    for(int i = 0; i < NUM_DATA; i++){
        a[i] = rand() % 10;
        b[i] = rand() % 10;
        c[i] = rand() % 10;
    }

    // vector sum on host
    timer.onTimer(4);
    
    for(int i = 0; i < NUM_DATA; i++)
        hd[i] = a[i] * b[i] / c[i];
    timer.offTimer(4);

    timer.onTimer(0);
    hipMalloc((void **)&da, memsize);
    hipMalloc((void **)&db, memsize);
    hipMalloc((void **)&dc, memsize);
    hipMalloc((void **)&dd, memsize);

    timer.onTimer(2);
    hipMemcpy(da, a, memsize, hipMemcpyHostToDevice);
    hipMemcpy(db, b, memsize, hipMemcpyHostToDevice);
    hipMemcpy(dc, c, memsize, hipMemcpyHostToDevice);
    timer.offTimer(2);

    timer.onTimer(1);
    int blockSize = 256;
    int gridSize = (NUM_DATA + blockSize - 1) / blockSize;
    vecMulDiv<<<gridSize, blockSize>>>(da, db, dc, dd, NUM_DATA);
    hipDeviceSynchronize();
    timer.offTimer(1);

    timer.onTimer(3);
    hipMemcpy(d, dd, memsize, hipMemcpyDeviceToHost);
    timer.offTimer(3);


    // problem 2

    // check results
    result = memcmp(d, hd, memsize);

    if (result ==0)
        printf("The data sum on the device (GPU) is the same as the data sum on the host (CPU)\n");
    else
        printf("The data sum on the device (GPU) is not the same as the data sum on the host (CPU)\n");

    // memory deallocation on host
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);

    free(a);
    free(b);
    free(c);
    free(d);
    free(hd);
    timer.offTimer(0);

    timer.printTimer();
    return 0;
}
